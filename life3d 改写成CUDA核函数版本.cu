/*-----------------------------------------------
 * 请在此处填写你的个人信息
 * 学号: SA24011134
 * 姓名: 李金优
 * 邮箱: ljinyou@mail.ustc.edu.cn
 ------------------------------------------------*/

#include <chrono>
#include <cstring>
#include <fstream>
#include <iostream>
#include <string>
#include "hip/hip_runtime.h" 



#define AT(x, y, z) universe[(x) * N * N + (y) * N + z]

using namespace std;
//using std::cin, std::cout, std::endl;
//using std::ifstream, std::ofstream;

// 存活细胞数
int population(int N, char *universe)
{
    int result = 0;
    for (int i = 0; i < N * N * N; i++)
        result += universe[i];
    return result;
}


void print_universe(int N, char *universe)
{
 
    if (N > 32)
        return;
    for (int x = 0; x < N; x++)
    {
        for (int y = 0; y < N; y++)
        {
            for (int z = 0; z < N; z++)
            {
                if (AT(x, y, z))
                    cout << "O ";
                else
                    cout << "* ";
            }
            cout << endl;
        }
        cout << endl;
    }
    cout << "population: " << population(N, universe) << endl;
}

//*********************************** 添加global
__global__ 
void life3d_run(int N, int cbsize,char *universe,char*next, int T)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z * blockDim.z + threadIdx.z;
  

        // outerloop: iter universe
        // inner loop: stencil
        int alive = 0;
        for (int dx = -1; dx <= 1; dx++)
            for (int dy = -1; dy <= 1; dy++)
                for (int dz = -1; dz <= 1; dz++)
                {
                    if (dx == 0 && dy == 0 && dz == 0)
                        continue;
                    int nx = (x + dx + N) % N;
                    int ny = (y + dy + N) % N;
                    int nz = (z + dz + N) % N;
                    alive += AT(nx, ny, nz);
                }
        
        if (AT(x, y, z) && (alive < 5 || alive > 7))
            next[x * N * N + y * N + z] = 0;
        else if (!AT(x, y, z) && alive == 6)
            next[x * N * N + y * N + z] = 1;
        else
            next[x * N * N + y * N + z] = AT(x, y, z); 
        //cudaMemcpy((void*)universe,(void*)universe,N*N*N,cudaMemcpyDeviceToDevice);





}
void check(int N, char *universe, int T)
{
    char *next = (char *)malloc(N * N * N);
    for (int t = 0; t < T; t++)
    {
        // outerloop: iter universe
        for (int x = 0; x < N; x++)
            for (int y = 0; y < N; y++)
                for (int z = 0; z < N; z++)
                {
                    // inner loop: stencil
                    int alive = 0;
                    for (int dx = -1; dx <= 1; dx++)
                        for (int dy = -1; dy <= 1; dy++)
                            for (int dz = -1; dz <= 1; dz++)
                            {
                                if (dx == 0 && dy == 0 && dz == 0)
                                    continue;
                                int nx = (x + dx + N) % N;
                                int ny = (y + dy + N) % N;
                                int nz = (z + dz + N) % N;
                                alive += AT(nx, ny, nz);
                            }
                    if (AT(x, y, z) && (alive < 5 || alive > 7))
                        next[x * N * N + y * N + z] = 0;
                    else if (!AT(x, y, z) && alive == 6)
                        next[x * N * N + y * N + z] = 1;
                    else
                        next[x * N * N + y * N + z] = AT(x, y, z);
                }
        memcpy(universe, next, N * N * N);
    }
    free(next);
}
// 读取输入文件
void read_file(char *input_file, char *buffer)
{
    ifstream file(input_file, std::ios::binary | std::ios::ate);
    if (!file.is_open())
    {
        cout << "Error: Could not open file " << input_file << std::endl;
        exit(1);
    }
    std::streamsize file_size = file.tellg();
    file.seekg(0, std::ios::beg);
    if (!file.read(buffer, file_size))
    {
        std::cerr << "Error: Could not read file " << input_file << std::endl;
        exit(1);
    }
    file.close();
}

// 写入输出文件
void write_file(char *output_file, char *buffer, int N)
{
    ofstream file(output_file, std::ios::binary | std::ios::trunc);
    if (!file)
    {
        cout << "Error: Could not open file " << output_file << std::endl;
        exit(1);
    }
    file.write(buffer, N * N * N);
    file.close();
}

int main(int argc, char **argv)
{
    // cmd args
    if (argc < 5)
    {
        cout << "usage: ./life3d N T input output" << endl;
        return 1;
    }
    int N = std::stoi(argv[1]);
    int T = std::stoi(argv[2]);
    char *input_file = argv[3];
    char *output_file = argv[4];


    char *universe = (char *)malloc(N * N * N);
    char *d_universe,*next;
    char*ans_universe = (char *)malloc(N * N * N);
    read_file(input_file, universe);

    memcpy(ans_universe, universe, N * N * N);
    check(N, ans_universe, T);
    int right_pop = population(N, ans_universe);


    int start_pop = population(N, universe);
    auto start_time = std::chrono::high_resolution_clock::now();
    
    

    //*********************************** 

    hipMalloc((void**)&d_universe,N*N*N);
    hipMemcpy((void*)d_universe,(void*)universe,N*N*N,hipMemcpyHostToDevice);
    hipMalloc((void**)&next,N*N*N);
    hipMemcpy((void*)next,(void*)universe,N*N*N,hipMemcpyHostToDevice);

     hipError_t err = hipGetLastError();

    if(err!= hipSuccess){
        printf("CUDA Error1: %s\n",hipGetErrorString(err));
    }
    
    int cbsize = 4;  // block size
    int gsize = (N+cbsize-1)/cbsize;
    dim3 blockSize(cbsize,cbsize,cbsize);
    dim3 gridSize(gsize,gsize,gsize);

   
    for(int i = 0;i<T;i++)
    {
        life3d_run<<<gridSize,blockSize>>>(N, cbsize,d_universe,next, T);
        hipDeviceSynchronize();
        hipMemcpy((void*)d_universe,(void*)next,N*N*N,hipMemcpyDeviceToDevice);
    }
    
    err = hipGetLastError();

    if(err!= hipSuccess){
        printf("CUDA Error2: %s\n",hipGetErrorString(err));
    }

    hipMemcpy((void*)universe,(void*)d_universe,N*N*N,hipMemcpyDeviceToHost);

      err = hipGetLastError();

    if(err!= hipSuccess){
        printf("CUDA Error3: %s\n",hipGetErrorString(err));
    }

    //*********************************** 

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end_time - start_time;
    int final_pop = population(N, universe);
    write_file(output_file, universe, N);

    cout << "start population: " << start_pop << endl;
    cout << "final population: " << final_pop << endl;
    cout << "right population: " << right_pop << endl;
    double time = duration.count();
    cout << "time: " << time << "s" << endl;
    cout << "cell per sec: " << T / time * N * N * N << endl;

    //*********************************** */
    hipFree(d_universe);

    free(universe);
    return 0;
}
